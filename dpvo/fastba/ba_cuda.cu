#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>
#include <memory>

#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Parallel.h>
#include "block_e.cuh"


#define GPU_1D_KERNEL_LOOP(i, n) \
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i<n; i += blockDim.x * gridDim.x)


#define NUM_THREADS 256
#define NUM_BLOCKS(batch_size) ((batch_size + NUM_THREADS - 1) / NUM_THREADS)

inline void release_assert(const char *file, int line, bool condition, const std::string &msg){
    if (!condition)
        throw std::runtime_error(std::string("Assertion failed: ") + file + " (" + std::to_string(line) + ")\n" + msg + "\n");
}

#define RASSERT(c) release_assert(__FILE__, __LINE__, c, "")
#define MRASSERT(c, m) release_assert(__FILE__, __LINE__, c, m)

void save(const char *filename, const torch::Tensor &data){
  const auto pickled = torch::pickle_save(data);
  std::ofstream fout(filename, std::ios::out | std::ios::binary);
  fout.write(pickled.data(), pickled.size());
  fout.close();
}

__device__ void
actSO3(const float *q, const float *X, float *Y) {
  float uv[3];
  uv[0] = 2.0 * (q[1]*X[2] - q[2]*X[1]);
  uv[1] = 2.0 * (q[2]*X[0] - q[0]*X[2]);
  uv[2] = 2.0 * (q[0]*X[1] - q[1]*X[0]);

  Y[0] = X[0] + q[3]*uv[0] + (q[1]*uv[2] - q[2]*uv[1]);
  Y[1] = X[1] + q[3]*uv[1] + (q[2]*uv[0] - q[0]*uv[2]);
  Y[2] = X[2] + q[3]*uv[2] + (q[0]*uv[1] - q[1]*uv[0]);
}

__device__  void
actSE3(const float *t, const float *q, const float *X, float *Y) {
  actSO3(q, X, Y);
  Y[3] = X[3];
  Y[0] += X[3] * t[0];
  Y[1] += X[3] * t[1];
  Y[2] += X[3] * t[2];
}

__device__ void
adjSE3(const float *t, const float *q, const float *X, float *Y) {
  float qinv[4] = {-q[0], -q[1], -q[2], q[3]};
  actSO3(qinv, &X[0], &Y[0]);
  actSO3(qinv, &X[3], &Y[3]);

  float u[3], v[3];
  u[0] = t[2]*X[1] - t[1]*X[2];
  u[1] = t[0]*X[2] - t[2]*X[0];
  u[2] = t[1]*X[0] - t[0]*X[1];

  actSO3(qinv, u, v);
  Y[3] += v[0];
  Y[4] += v[1];
  Y[5] += v[2];
}

__device__ void 
relSE3(const float *ti, const float *qi, const float *tj, const float *qj, float *tij, float *qij) {
  qij[0] = -qj[3] * qi[0] + qj[0] * qi[3] - qj[1] * qi[2] + qj[2] * qi[1],
  qij[1] = -qj[3] * qi[1] + qj[1] * qi[3] - qj[2] * qi[0] + qj[0] * qi[2],
  qij[2] = -qj[3] * qi[2] + qj[2] * qi[3] - qj[0] * qi[1] + qj[1] * qi[0],
  qij[3] =  qj[3] * qi[3] + qj[0] * qi[0] + qj[1] * qi[1] + qj[2] * qi[2],

  actSO3(qij, ti, tij);
  tij[0] = tj[0] - tij[0];
  tij[1] = tj[1] - tij[1];
  tij[2] = tj[2] - tij[2];
}

  
__device__ void
expSO3(const float *phi, float* q) {
  // SO3 exponential map
  float theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  float theta_p4 = theta_sq * theta_sq;

  float theta = sqrtf(theta_sq);
  float imag, real;

  if (theta_sq < 1e-8) {
    imag = 0.5 - (1.0/48.0)*theta_sq + (1.0/3840.0)*theta_p4;
    real = 1.0 - (1.0/ 8.0)*theta_sq + (1.0/ 384.0)*theta_p4;
  } else {
    imag = sinf(0.5 * theta) / theta;
    real = cosf(0.5 * theta);
  }

  q[0] = imag * phi[0];
  q[1] = imag * phi[1];
  q[2] = imag * phi[2];
  q[3] = real;

}

__device__ void
crossInplace(const float* a, float *b) {
  float x[3] = {
    a[1]*b[2] - a[2]*b[1],
    a[2]*b[0] - a[0]*b[2],
    a[0]*b[1] - a[1]*b[0], 
  };

  b[0] = x[0];
  b[1] = x[1];
  b[2] = x[2];
}

__device__ void
expSE3(const float *xi, float* t, float* q) {
  // SE3 exponential map

  expSO3(xi + 3, q);
  float tau[3] = {xi[0], xi[1], xi[2]};
  float phi[3] = {xi[3], xi[4], xi[5]};

  float theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  float theta = sqrtf(theta_sq);

  t[0] = tau[0]; 
  t[1] = tau[1]; 
  t[2] = tau[2];

  if (theta > 1e-4) {
    float a = (1 - cosf(theta)) / theta_sq;
    crossInplace(phi, tau);
    t[0] += a * tau[0];
    t[1] += a * tau[1];
    t[2] += a * tau[2];

    float b = (theta - sinf(theta)) / (theta * theta_sq);
    crossInplace(phi, tau);
    t[0] += b * tau[0];
    t[1] += b * tau[1];
    t[2] += b * tau[2];
  }
}


__device__ void
retrSE3(const float *xi, const float* t, const float* q, float* t1, float* q1) {
  // retraction on SE3 manifold

  float dt[3] = {0, 0, 0};
  float dq[4] = {0, 0, 0, 1};
  
  expSE3(xi, dt, dq);

  q1[0] = dq[3] * q[0] + dq[0] * q[3] + dq[1] * q[2] - dq[2] * q[1];
  q1[1] = dq[3] * q[1] + dq[1] * q[3] + dq[2] * q[0] - dq[0] * q[2];
  q1[2] = dq[3] * q[2] + dq[2] * q[3] + dq[0] * q[1] - dq[1] * q[0];
  q1[3] = dq[3] * q[3] - dq[0] * q[0] - dq[1] * q[1] - dq[2] * q[2];

  actSO3(dq, t, t1);
  t1[0] += dt[0];
  t1[1] += dt[1];
  t1[2] += dt[2];
}



__global__ void pose_retr_kernel(const int t0, const int t1,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    torch::PackedTensorAccessor32<mtype,2,torch::RestrictPtrTraits> update)
{
  GPU_1D_KERNEL_LOOP(i, t1 - t0) {
    const float t = t0 + i;
    float t1[3], t0[3] = { poses[t][0], poses[t][1], poses[t][2] };
    float q1[4], q0[4] = { poses[t][3], poses[t][4], poses[t][5], poses[t][6] };

    float xi[6] = {
      update[i][0],
      update[i][1],
      update[i][2],
      update[i][3],
      update[i][4],
      update[i][5],
    };

    retrSE3(xi, t0, q0, t1, q1);

    poses[t][0] = t1[0];
    poses[t][1] = t1[1];
    poses[t][2] = t1[2];
    poses[t][3] = q1[0];
    poses[t][4] = q1[1];
    poses[t][5] = q1[2];
    poses[t][6] = q1[3];
  }
}


__global__ void patch_retr_kernel(
    torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> index,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> patches,
    torch::PackedTensorAccessor32<mtype,1,torch::RestrictPtrTraits> update)
{
  GPU_1D_KERNEL_LOOP(n, index.size(0)) {
    const int p = patches.size(2);
    const int ix = index[n];
  
    float d = patches[ix][2][0][0];
    d = d + update[n];
    d = (d > 20) ? 1.0 : d;
    d = max(d, 1e-4);

    for (int i=0; i<p; i++) {
      for (int j=0; j<p; j++) {
        patches[ix][2][i][j] = d;
      }
    }
  }
}


__global__ void reprojection_residuals_and_hessian(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> patches,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> target,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> weight,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> lmbda,
    const torch::PackedTensorAccessor32<long,2,torch::RestrictPtrTraits> ij_xself,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> kk,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ku,
    torch::PackedTensorAccessor32<double,1,torch::RestrictPtrTraits> r_total,
    torch::PackedTensorAccessor32<mtype,3,torch::RestrictPtrTraits> E_lookup,
    torch::PackedTensorAccessor32<mtype,2,torch::RestrictPtrTraits> B,
    torch::PackedTensorAccessor32<mtype,2,torch::RestrictPtrTraits> E,
    torch::PackedTensorAccessor32<mtype,1,torch::RestrictPtrTraits> C,
    torch::PackedTensorAccessor32<mtype,1,torch::RestrictPtrTraits> v,
    torch::PackedTensorAccessor32<mtype,1,torch::RestrictPtrTraits> u, const int t0, const int ppf)
{

  __shared__ float fx, fy, cx, cy;
  if (threadIdx.x == 0) {
    fx = intrinsics[0][0];
    fy = intrinsics[0][1];
    cx = intrinsics[0][2];
    cy = intrinsics[0][3];
  }

  bool eff_impl = (ppf > 0);

  __syncthreads();

  GPU_1D_KERNEL_LOOP(n, ii.size(0)) {
    int k = ku[n]; // inverse indices
    int ix = ii[n];
    int jx = jj[n];
    int kx = kk[n]; // actual
    int ijx, ijs;
    if (eff_impl){
      ijx = ij_xself[0][n];
      ijs = ij_xself[1][n];
    }

    float ti[3] = { poses[ix][0], poses[ix][1], poses[ix][2] };
    float tj[3] = { poses[jx][0], poses[jx][1], poses[jx][2] };
    float qi[4] = { poses[ix][3], poses[ix][4], poses[ix][5], poses[ix][6] };
    float qj[4] = { poses[jx][3], poses[jx][4], poses[jx][5], poses[jx][6] };

    float Xi[4], Xj[4];
    Xi[0] = (patches[kx][0][1][1] - cx) / fx;
    Xi[1] = (patches[kx][1][1][1] - cy) / fy;
    Xi[2] = 1.0;
    Xi[3] = patches[kx][2][1][1];
    
    float tij[3], qij[4];
    relSE3(ti, qi, tj, qj, tij, qij);
    actSE3(tij, qij, Xi, Xj);

    const float X = Xj[0];
    const float Y = Xj[1];
    const float Z = Xj[2];
    const float W = Xj[3];

    const float d = (Z >= 0.2) ? 1.0 / Z : 0.0; 
    const float d2 = d * d;

    const float x1 = fx * (X / Z) + cx;
    const float y1 = fy * (Y / Z) + cy;

    const float rx = target[n][0] - x1;
    const float ry = target[n][1] - y1;

    const bool in_bounds = (sqrt(rx*rx + ry*ry) < 128) && (Z > 0.2) &&
      (x1 > -64) && (y1 > -64) && (x1 < 2*cx + 64) && (y1 < 2*cy + 64);

    const float mask = in_bounds ? 1.0 : 0.0;

    ix = ix - t0;
    jx = jx - t0;

    for (int row=0; row<2; row++) {

      float *Jj, Ji[6], Jz, r, w;

      if (row == 0){

        r = target[n][0] - x1;
        w = mask * weight[n][0];

        Jz = fx * (tij[0] * d - tij[2] * (X * d2));
        Jj = (float[6]){fx*W*d, 0, fx*-X*W*d2, fx*-X*Y*d2, fx*(1+X*X*d2), fx*-Y*d};

      } else {

        r = target[n][1] - y1;
        w = mask * weight[n][1];

        Jz = fy * (tij[1] * d - tij[2] * (Y * d2));
        Jj = (float[6]){0, fy*W*d, fy*-Y*W*d2, fy*(-1-Y*Y*d2), fy*(X*Y*d2), fy*X*d};

      }

      atomicAdd(&r_total[0],  w * r * r);

      adjSE3(tij, qij, Jj, Ji);

      for (int i=0; i<6; i++) {
        for (int j=0; j<6; j++) {
          if (ix >= 0)
            atomicAdd(&B[6*ix+i][6*ix+j],  w * Ji[i] * Ji[j]);
          if (jx >= 0)
            atomicAdd(&B[6*jx+i][6*jx+j],  w * Jj[i] * Jj[j]);
          if (ix >= 0 && jx >= 0) {
            atomicAdd(&B[6*ix+i][6*jx+j], -w * Ji[i] * Jj[j]);
            atomicAdd(&B[6*jx+i][6*ix+j], -w * Jj[i] * Ji[j]);
          }
        }
      }

      for (int i=0; i<6; i++) {
        if (eff_impl){
          atomicAdd(&E_lookup[ijs][kx % ppf][i],  -w * Jz * Ji[i]);
          atomicAdd(&E_lookup[ijx][kx % ppf][i],  w * Jz * Jj[i]);
        } else {
          if (ix >= 0)
            atomicAdd(&E[6*ix+i][k], -w * Jz * Ji[i]);
          if (jx >= 0)
            atomicAdd(&E[6*jx+i][k],  w * Jz * Jj[i]);
        }

      }

      for (int i=0; i<6; i++) {
        if (ix >= 0)
          atomicAdd(&v[6*ix+i], -w * r * Ji[i]);
        if (jx >= 0)
          atomicAdd(&v[6*jx+i],  w * r * Jj[i]);
      }

      atomicAdd(&C[k], w * Jz * Jz);
      atomicAdd(&u[k], w *  r * Jz);
    }
  }
}


__global__ void reproject(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> patches,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> kk,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> coords) {

  __shared__ float fx, fy, cx, cy;
  if (threadIdx.x == 0) {
    fx = intrinsics[0][0];
    fy = intrinsics[0][1];
    cx = intrinsics[0][2];
    cy = intrinsics[0][3];
  }

  __syncthreads();

  GPU_1D_KERNEL_LOOP(n, ii.size(0)) {
    int ix = ii[n];
    int jx = jj[n];
    int kx = kk[n];

    float ti[3] = { poses[ix][0], poses[ix][1], poses[ix][2] };
    float tj[3] = { poses[jx][0], poses[jx][1], poses[jx][2] };
    float qi[4] = { poses[ix][3], poses[ix][4], poses[ix][5], poses[ix][6] };
    float qj[4] = { poses[jx][3], poses[jx][4], poses[jx][5], poses[jx][6] };

    float tij[3], qij[4];
    relSE3(ti, qi, tj, qj, tij, qij);

    float Xi[4], Xj[4];
    for (int i=0; i<patches.size(2); i++) {
      for (int j=0; j<patches.size(3); j++) {
        
        Xi[0] = (patches[kx][0][i][j] - cx) / fx;
        Xi[1] = (patches[kx][1][i][j] - cy) / fy;
        Xi[2] = 1.0;
        Xi[3] = patches[kx][2][i][j];

        actSE3(tij, qij, Xi, Xj);

        coords[n][0][i][j] = fx * (Xj[0] / Xj[2]) + cx;
        coords[n][1][i][j] = fy * (Xj[1] / Xj[2]) + cy;
        // coords[n][2][i][j] = 1.0 / Xj[2];

      }
    }
  }
}



std::vector<torch::Tensor> cuda_ba(
    torch::Tensor poses,
    torch::Tensor patches,
    torch::Tensor intrinsics,
    torch::Tensor target,
    torch::Tensor weight,
    torch::Tensor lmbda,
    torch::Tensor ii,
    torch::Tensor jj,
    torch::Tensor kk,
    const int PPF,
    const int t0, const int t1, const int iterations, bool eff_impl)
{

  auto ktuple = torch::_unique(kk, true, true);
  torch::Tensor kx = std::get<0>(ktuple);
  torch::Tensor ku = std::get<1>(ktuple);

  const int N = t1 - t0;    // number of poses
  const int M = kx.size(0); // number of patches
  const int P = patches.size(3); // patch size

  // auto opts = torch::TensorOptions()
  //   .dtype(torch::kFloat32).device(torch::kCUDA);

  poses = poses.view({-1, 7});
  patches = patches.view({-1,3,P,P});
  intrinsics = intrinsics.view({-1, 4});

  target = target.view({-1, 2});
  weight = weight.view({-1, 2});

  const int num = ii.size(0);
  torch::Tensor B = torch::empty({6*N, 6*N}, mdtype);
  torch::Tensor E = torch::empty({0, 0}, mdtype);
  torch::Tensor C = torch::empty({M}, mdtype);

  torch::Tensor v = torch::empty({6*N}, mdtype);
  torch::Tensor u = torch::empty({1*M}, mdtype);

  torch::Tensor r_total = torch::empty({1}, torch::dtype(torch::kFloat64).device(torch::kCUDA));

  auto blockE = std::make_unique<EfficentE>();

  if (eff_impl)
    blockE = std::make_unique<EfficentE>(ii, jj, kx, PPF, t0);
  else
    E = torch::empty({6*N, 1*M}, mdtype);

  for (int itr=0; itr < iterations; itr++) {

    B.zero_();
    E.zero_();
    C.zero_();
    v.zero_();
    u.zero_();
    r_total.zero_();
    blockE->E_lookup.zero_();

    v = v.view({6*N});
    u = u.view({1*M});

    reprojection_residuals_and_hessian<<<NUM_BLOCKS(ii.size(0)), NUM_THREADS>>>(
      poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      patches.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
      intrinsics.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      target.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      weight.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      lmbda.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
      blockE->ij_xself.packed_accessor32<long,2,torch::RestrictPtrTraits>(),
      ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      kk.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      ku.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      r_total.packed_accessor32<double,1,torch::RestrictPtrTraits>(),
      blockE->E_lookup.packed_accessor32<mtype,3,torch::RestrictPtrTraits>(),
      B.packed_accessor32<mtype,2,torch::RestrictPtrTraits>(),
      E.packed_accessor32<mtype,2,torch::RestrictPtrTraits>(),
      C.packed_accessor32<mtype,1,torch::RestrictPtrTraits>(),
      v.packed_accessor32<mtype,1,torch::RestrictPtrTraits>(),
      u.packed_accessor32<mtype,1,torch::RestrictPtrTraits>(), t0, blockE->ppf);

    // std::cout << "Total residuals: " << r_total.item<double>() << std::endl;
    v = v.view({6*N, 1});
    u = u.view({1*M, 1});

    torch::Tensor Q = 1.0 / (C + lmbda).view({1, M});

    if (t1 - t0 == 0) {

      torch::Tensor Qt = torch::transpose(Q, 0, 1);
      torch::Tensor dZ = Qt * u;

      dZ = dZ.view({M});

      patch_retr_kernel<<<NUM_BLOCKS(M), NUM_THREADS>>>(
        kx.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
        patches.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
        dZ.packed_accessor32<mtype,1,torch::RestrictPtrTraits>());

    }  else {

      torch::Tensor dX, dZ, Qt = torch::transpose(Q, 0, 1);
      torch::Tensor I = torch::eye(6*N, mdtype);

      if (eff_impl) {

        torch::Tensor EQEt = blockE->computeEQEt(N, Q);
        torch::Tensor EQu = blockE->computeEv(N, Qt * u);

        torch::Tensor S = B - EQEt;
        torch::Tensor y = v - EQu;

        S += I * (1e-4 * S + 1.0);
        torch::Tensor U = std::get<0>(at::linalg_cholesky_ex(S));
        dX = torch::cholesky_solve(y, U);
        torch::Tensor EtdX = blockE->computeEtv(M, dX);
        dZ = Qt * (u - EtdX);

      } else {

        torch::Tensor EQ = E * Q;
        torch::Tensor Et = torch::transpose(E, 0, 1);

        torch::Tensor S = B - torch::matmul(EQ, Et);
        torch::Tensor y = v - torch::matmul(EQ,  u);

        S += I * (1e-4 * S + 1.0);
        torch::Tensor U = std::get<0>(at::linalg_cholesky_ex(S));
        dX = torch::cholesky_solve(y, U);
        dZ = Qt * (u - torch::matmul(Et, dX));

      }

      dX = dX.view({N, 6});
      dZ = dZ.view({M});

      pose_retr_kernel<<<NUM_BLOCKS(N), NUM_THREADS>>>(t0, t1,
          poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
          dX.packed_accessor32<mtype,2,torch::RestrictPtrTraits>());

      patch_retr_kernel<<<NUM_BLOCKS(M), NUM_THREADS>>>(
          kx.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
          patches.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
          dZ.packed_accessor32<mtype,1,torch::RestrictPtrTraits>());
    }
  }
  
  return {};
}


torch::Tensor cuda_reproject(
    torch::Tensor poses,
    torch::Tensor patches,
    torch::Tensor intrinsics,
    torch::Tensor ii,
    torch::Tensor jj, 
    torch::Tensor kk)
{

  const int N = ii.size(0);
  const int P = patches.size(3); // patch size

  poses = poses.view({-1, 7});
  patches = patches.view({-1,3,P,P});
  intrinsics = intrinsics.view({-1, 4});

  auto opts = torch::TensorOptions()
    .dtype(torch::kFloat32).device(torch::kCUDA);

  torch::Tensor coords = torch::empty({N, 2, P, P}, opts);

  reproject<<<NUM_BLOCKS(N), NUM_THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    patches.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    kk.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    coords.packed_accessor32<float,4,torch::RestrictPtrTraits>());

  return coords.view({1, N, 2, P, P});

}